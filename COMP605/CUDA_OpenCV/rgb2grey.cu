#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <iostream>
#include <string>

#include <sys/stat.h>
#include <sys/time.h>

#include <opencv2/opencv.hpp>
#include <opencv2/imgcodecs.hpp>
#include <hip/hip_runtime.h>


using namespace cv;
using namespace std;
#define BLOCK_SIZE 32

__global__ void RGB2GREY(uchar4 *image_RGB, unsigned char *image_GREY, int cols, int rows);

int main(int argc, char** argv)
{
    string image_name = "bear.jpg";
    string image_path = "/home/perimbeti/COMP605_HW4/bear.jpg";

    // Reading in the input image file ("bear.jpg")
    cv::Mat image = imread(image_path.c_str(), IMREAD_COLOR);
    if (image.empty())
    {
        cerr << "[ERROR] Couldn't open file: " << image_path << endl;
              return 1;
    }

    cv::Mat image_RGB;
    cv::cvtColor(image, image_RGB, cv::COLOR_BGR2RGBA);

    uint num_cols = image.cols; // image width
    uint num_rows = image.rows; // image height

    hipEvent_t start, end;
    float time_elapsed;

    int i = 0;  // used for kernel call repitions for time differences

    unsigned char *host_output;
    unsigned char *dim_output;
    uchar4 *dim_input;

    printf("\n");
    cout << "----------------------------------------------------------------" << endl;
    cout << " This program uses CUDA to convert an image in RGB to Greyscale" << endl;
    cout << "----------------------------------------------------------------" << endl;
    cout << "              Image Title: bear.jpg                     " << endl;
    cout << "   Image Dimenstions (rows x cols) = " << num_rows << " x " << num_cols << endl;
    cout << "----------------------------------------------------------------" << endl;

    int input_size = num_cols * num_rows * sizeof(uchar4);
    int output_size = num_cols * num_rows * sizeof(unsigned char);

    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid(ceil(num_cols/(float)dimBlock.x) ,ceil(num_rows/(float)dimBlock.y));

    cout << "             GRID SIZE (Y x X) = " << dimGrid.y << " x "  << dimGrid.x <<endl;
    cout << "            BLOCK SIZE (N x N) = " << dimBlock.x << " x " << dimBlock.y <<endl;
    cout << "----------------------------------------------------------------" << endl;

    // Starting the Timer
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    // Memory Allocation for RGB Input and Output Image GPU
    hipMalloc((void**)&dim_input, input_size);
    hipMalloc((void**)&dim_output, output_size);

    // Memory Allocation for output greyscale image on CPU
    host_output = (unsigned char *)malloc(output_size);

    uchar4 *host_input = (uchar4 *)image_RGB.ptr<unsigned char>(0); // Conversion to 1D array

    hipMemcpy(dim_input, host_input, input_size, hipMemcpyHostToDevice); // Copies the input image to the device

    // CALL to Conversion KERNEL - looped 1000 times for execution time differences

    for(i = 0; i < 1000; i++) {
        RGB2GREY<<<dimGrid,dimBlock>>>(dim_input, dim_output, num_cols, num_rows);
    }

    // Copy output image results to CPU
    hipMemcpy(host_output, dim_output,output_size, hipMemcpyDeviceToHost);

    /*  --- STOP THE TIMER  -- */
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time_elapsed, start, end);

    cout << "                   Time Elapsed: " << time_elapsed << " ms  "  << endl;
    cout << "----------------------------------------------------------------" << endl;
    cout << "            Output Image Title: greyscale_bear.jpg     " << endl;
    cout << "----------------------------------------------------------------" << endl;
    printf("\n");

    // Writing greyscale image to output file (""greyscale_bear.jpg)
    cv::Mat output_image = Mat(num_rows, num_cols, CV_8UC1, host_output);
    imwrite("greyscale_"+image_name, output_image);

    //Free all allocated memory
    hipFree(dim_input);
    hipFree(dim_output);
    free(host_output);

    return 0;
}

// Kernel call for Image RGB to Greyscale conversion

__global__ void RGB2GREY(uchar4 *image_RGB, unsigned char *image_GREY, int cols, int rows){
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if( x >= cols || y >= rows) {return;} // If dimensions do not match up

  uchar4 rgb = image_RGB[x + y * cols ];
  unsigned char lum =  (0.30f * rgb.x) + (0.59f * rgb.y) + (0.11f * rgb.z); // luminosity effect
  image_GREY[x + y * cols] = lum;

}
